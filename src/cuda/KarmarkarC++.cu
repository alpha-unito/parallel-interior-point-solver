#include "hip/hip_runtime.h"
#include "hipsolver.h"
#include "myutilsCUDA.h"
#include <hipblas.h>

constexpr size_type Nblock  = 1024;
constexpr size_type Nthread = 256;

constexpr data_type ONE  = 1.0;
constexpr data_type MONE = -1.0;
constexpr data_type ZERO = 0.0;

/*
    x_i=x_i*y_i
*/
__global__ void sym_rank1(data_type *inout, const data_type *in, const size_type N) {

    int xid = threadIdx.x + (blockDim.x * blockIdx.x);

    while (xid < N) {
        inout[xid] = inout[xid] * in[xid];
        xid += blockDim.x * gridDim.x;
    }
}
/*
    A_ij=A_ij*x_j
*/
__global__ void sym_rank2(data_type *A, const data_type *x, const size_type Nrow,
                          const size_type Ncol) {

    std::size_t tid = blockIdx.x;
    // xid<Ncol

    while (tid < Nrow) {
        std::size_t xid = threadIdx.x;
        while (xid < Ncol) {
            A[tid * Ncol + xid] = A[tid * Ncol + xid] * x[xid];
            xid += blockDim.x;
        }
        tid += gridDim.x;
    }
}

__global__ void copy(data_type *A, data_type *B, const size_type N) {
    int xid = threadIdx.x + (blockDim.x * blockIdx.x);

    while (xid < N) {
        B[xid] = A[xid];
        xid += blockDim.x * gridDim.x;
    }
}

__global__ void dsum(data_type *A, data_type y, const size_type N) {
    int xid = threadIdx.x + (blockDim.x * blockIdx.x);

    while (xid < N) {
        A[xid] = y;
        xid += blockDim.x * gridDim.x;
    }
}

void Kamarkar(std::vector<data_type> &A_vec, std::vector<data_type> &B_vec,
              std::vector<data_type> &BB_vec, std::vector<data_type> &c_vec,
              std::vector<data_type> &c_vec1, std::vector<data_type> &x_vec,
              std::vector<data_type> &w_vec, std::vector<data_type> &w_vec1,
              std::vector<data_type> &d_vec, std::vector<data_type> &y_vec, const size_type Nrow,
              const size_type Ncol, const size_type N, const size_type N1, const data_type alpha,
              const int max_iter, const double tol) {

    std::chrono::time_point<std::chrono::high_resolution_clock> start;
    std::chrono::time_point<std::chrono::high_resolution_clock> end;

    double *A_vec_dev;
    double *B_vec_dev;
    double *BB_vec_dev;
    double *c_vec_dev;
    double *co_vec_dev;
    double *c_vec1_dev;
    double *x_vec_dev;
    double *w_vec_dev;
    double *w_vec1_dev;
    double *d_vec_dev;
    double *y_vec_dev;

    double norm{0.0};
    double norm_dot{0.0};
    double res{1.0};

    hipblasHandle_t handle;
    hipsolverHandle_t cuSolverHandle;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;

    hipsolverDnCreate(&cuSolverHandle);
    hipblasCreate(&handle);

    hipMalloc(&A_vec_dev, sizeof(data_type) * N);
    hipMalloc(&B_vec_dev, sizeof(data_type) * N1);
    hipMalloc(&BB_vec_dev, sizeof(data_type) * (Nrow + 1) * (Nrow + 1));
    hipMalloc(&c_vec_dev, sizeof(data_type) * Ncol);
    hipMalloc(&co_vec_dev, sizeof(data_type) * Ncol);
    hipMalloc(&c_vec1_dev, sizeof(data_type) * (Nrow + 1));
    hipMalloc(&x_vec_dev, sizeof(data_type) * Ncol);
    hipMalloc(&w_vec_dev, sizeof(data_type) * (Nrow + 1));
    hipMalloc(&w_vec1_dev, sizeof(data_type) * Ncol);
    hipMalloc(&d_vec_dev, sizeof(data_type) * Ncol);
    hipMalloc(&y_vec_dev, sizeof(data_type) * Ncol);

    hipMemcpy(A_vec_dev, A_vec.data(), sizeof(data_type) * N, hipMemcpyHostToDevice);
    hipMemcpy(c_vec_dev, c_vec.data(), sizeof(data_type) * Ncol, hipMemcpyHostToDevice);
    hipMemcpy(x_vec_dev, x_vec.data(), sizeof(data_type) * Ncol, hipMemcpyHostToDevice);

    hipMemcpy(co_vec_dev, c_vec_dev, sizeof(double) * (Ncol), hipMemcpyDeviceToDevice);

    for (int j = 0; j < Ncol; ++j) {
        B_vec[Nrow * Ncol + j] = 1.0;
    }

    hipMemcpy(B_vec_dev, B_vec.data(), sizeof(data_type) * N1, hipMemcpyHostToDevice);

    int iter = 0;
    int m    = Nrow + 1;
    int n    = Nrow + 1;
    int k    = Ncol;

    while (res > tol) {

        start = std::chrono::high_resolution_clock::now();

        norm     = 0.0;
        norm_dot = 0.0;

        sym_rank1<<<Nblock, Nthread>>>(c_vec_dev, x_vec_dev, Ncol);

        sym_rank2<<<Nblock, Nthread>>>(A_vec_dev, x_vec_dev, Nrow, Ncol);

        hipMemcpy(B_vec_dev, A_vec_dev, sizeof(double) * (N), hipMemcpyDeviceToDevice);

        // c1=B*c
        hipblasDgemv(handle, HIPBLAS_OP_T, Ncol, Nrow + 1, &ONE, B_vec_dev, Ncol, c_vec_dev, 1, &ZERO,
                    c_vec1_dev, 1);

        // B*BT, B=Nrow+1 x Ncol, BT=Ncol x Nrow+1       m=Nrow+1, n=Ncol,k=Ncol

        hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, &ONE, B_vec_dev, k, B_vec_dev, k,
                    &ZERO, BB_vec_dev, m);

        int bufferSize = 0;
        int *info      = NULL;
        double *buffer = NULL;
        int h_info     = 0;

        hipsolverDnDpotrf_bufferSize(cuSolverHandle, uplo, n, BB_vec_dev, n, &bufferSize);

        hipMalloc(&buffer, sizeof(double) * bufferSize);
        hipMalloc(&info, sizeof(int));


        hipsolverDnDpotrf(cuSolverHandle, uplo, n, BB_vec_dev, n, buffer, bufferSize, info);

        hipMemcpy(&h_info, info, sizeof(int), hipMemcpyDeviceToHost);
        if (0 != h_info) {
            fprintf(stderr, "Error: Cholesky factorization failed\n");
            std::cout << h_info << std::endl;
            exit(100);
        }

        hipMemcpy(w_vec_dev, c_vec1_dev, sizeof(double) * (Nrow + 1), hipMemcpyDeviceToDevice);

        hipsolverDnDpotrs(cuSolverHandle, uplo, n, 1, BB_vec_dev, n, w_vec_dev, n, info);

        hipblasDgemv(handle, HIPBLAS_OP_N, Ncol, Nrow + 1, &MONE, B_vec_dev, Ncol, w_vec_dev, 1, &ONE,
                    c_vec_dev, 1);

        hipblasDnrm2(handle, Ncol, c_vec_dev, 1, &norm); // euclidean norm

        double tmp = -alpha / norm;

        dsum<<<Nblock, Nthread>>>(d_vec_dev, 1.0 / Ncol, Ncol);

        hipblasDaxpy(handle, Ncol, &tmp, c_vec_dev, 1, d_vec_dev, 1);

        hipblasDdot(handle, Ncol, x_vec_dev, 1, d_vec_dev, 1, &norm_dot);

        sym_rank1<<<Nblock, Nthread>>>(x_vec_dev, d_vec_dev, Ncol);

        norm_dot = 1 / norm_dot;
        hipblasDscal(handle, Ncol, &norm_dot, x_vec_dev, 1);

        hipblasDdot(handle, Ncol, co_vec_dev, 1, x_vec_dev, 1, &res);

        res = std::abs(res);

        hipMemcpy(x_vec.data(), x_vec_dev, sizeof(data_type) * (Ncol), hipMemcpyDeviceToHost);

        end = std::chrono::high_resolution_clock::now();

        std::cout << "[Iter " << iter << "] norm " << norm << " norm dot " << 1 / norm_dot << " res "
                  << res << " Iter Time: " << std::chrono::duration<double>(end - start).count()
                  << std::endl;
        // print_vector(x_vec,Ncol);

        if (++iter > max_iter) {
            std::cout << "Reached max iterations iter=" << iter << std::endl;
            break;
        }
    }
}
